#include "hip/hip_runtime.h"
/*INF1029 - INT ARQUITETURA COMPUTADORES - 2022.2 - 3WA
Trabalho 4 - Implementação do algoritmo otimizado para o produto de matrizes
Nome: Eric Leão     Matrícula: 2110694
Nome: Pedro Machado Peçanha    Matrícula: 2110535*/


#include <assert.h>
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <stdio.h>
#include <stdlib.h>
#include "matrix_lib.h"
extern "C" {
#include "timer.h"
}

#define PARTIAL 0
#define COMPLETE 1
#define ERROR -1
#define MIBSIZE 1048576 

void fill_matrix(struct matrix *matrixX, FILE *arquivoBinX);

void fill_binary(struct matrix *matrixX, FILE *arquivoBinX);

void print_matrix(struct matrix matrix, char matrixChar);
int main(int argc, char *argv[]) {
  int height1 = atoi(argv[2]), height2 = atoi(argv[4]), width1 = atoi(argv[3]),
      width2 = atoi(argv[5]), n_threads = atoi(argv[6]),
      n_blocks = atoi(argv[7]), memoria = atoi(argv[8]);
  hipError_t hipError_t;
  FILE *arquivoBin1, *arquivoBin2;
  float scalar = strtof(argv[1], NULL), diferencaTotal;
  //  float *matrixRows;
  char *filename1 = argv[9], *filename2 = argv[10], *filename3 = argv[11],
       *filename4 = argv[12];
  struct timeval overall_t1, overall_t2, start, stop;
  // Mark overall start time
  gettimeofday(&overall_t1, NULL);
  struct matrix matrix1, matrix2, matrix3;
  if (!set_grid_size(n_threads, n_blocks)) {
    perror("Valores default serão utilizados.");
   
  }

  int tipo = COMPLETE;

  if (((width1 * sizeof(float)) + (height2 * width2 * sizeof(float)) +
  (width2 * sizeof(float))) > memoria * MIBSIZE) {
// completa
printf("Matrix parcial não cabe.\nTamanho completo das matrizes (em bytes): "
      "%ld\nTamanho da memória da GPU: %d\n",
      (width1 * sizeof(float)) +
          (height2 * width2 * sizeof(float)) +
          (width2 * sizeof(float)),
      memoria * MIBSIZE);
tipo = ERROR;
}

  else if (((height1 * width1 * sizeof(float)) + (height2 * width2 * sizeof(float)) +
       (height1 * width2 * sizeof(float))) > memoria * MIBSIZE) {
    // completa
    printf("Matrix completa não cabe.\nTamanho completo das matrizes (em bytes): "
           "%ld\nTamanho da memória da GPU: %d\n",
           (height1 * width1 * sizeof(float)) +
               (height2 * width2 * sizeof(float)) +
               (height1 * width2 * sizeof(float)),
           memoria * MIBSIZE);
    tipo = PARTIAL;
  }

  matrix1.alloc_mode = tipo;
  matrix2.alloc_mode = tipo;
  matrix3.alloc_mode = tipo;


  // ARMAZENAMENTO LOCAL

  // matrix 1
  {
    matrix1.height = height1;
    matrix1.width = width1;
    matrix1.h_rows =
        (float *)aligned_alloc(32, sizeof(float) * height1 * width1);
    assert(matrix1.h_rows);
  }
  // matrix 2
  {
    matrix2.height = height2;
    matrix2.width = width2;
    matrix2.h_rows =
        (float *)aligned_alloc(32, sizeof(float) * height2 * width2);
    assert(matrix2.h_rows);
  }
  // matrix 3
  {
    matrix3.height = height1;
    matrix3.width = width2;
    matrix3.h_rows =
        (float *)aligned_alloc(32, sizeof(float) * height1 * width2);
    assert(matrix3.h_rows);
  }
  // ARMAZENAMENTO NA GPU
  if (tipo == COMPLETE) {
    hipError_t = hipMalloc(&matrix1.d_rows, sizeof(float) * height1 * width1);
    if (hipError_t != hipSuccess) {
      printf("hipMalloc d_x returned error %s (code %d)\n",
             hipGetErrorString(hipError_t), hipError_t);
      return 1;
    }
    hipError_t = hipMalloc(&matrix2.d_rows, sizeof(float) * height2 * width2);
    if (hipError_t != hipSuccess) {
      printf("hipMalloc d_x returned error %s (code %d)\n",
             hipGetErrorString(hipError_t), hipError_t);
      return 1;
    }
    hipError_t = hipMalloc(&matrix3.d_rows, sizeof(float) * height1 * width2);
    if (hipError_t != hipSuccess) {
      printf("hipMalloc d_x returned error %s (code %d)\n",
             hipGetErrorString(hipError_t), hipError_t);
      return 1;
    }
  }

  else if (tipo == PARTIAL){
    hipError_t = hipMalloc(&matrix1.d_rows, sizeof(float) *width1);
    if (hipError_t != hipSuccess) {
      printf("hipMalloc d_x returned error %s (code %d)\n",
             hipGetErrorString(hipError_t), hipError_t);
      return 1;
    }
    hipError_t = hipMalloc(&matrix2.d_rows, sizeof(float) * height2 * width2);
    if (hipError_t != hipSuccess) {
      printf("hipMalloc d_x returned error %s (code %d)\n",
             hipGetErrorString(hipError_t), hipError_t);
      return 1;
    }
    hipError_t = hipMalloc(&matrix3.d_rows, sizeof(float) *width2);
    if (hipError_t != hipSuccess) {
      printf("hipMalloc d_x returned error %s (code %d)\n",
             hipGetErrorString(hipError_t), hipError_t);
      return 1;
    }
  }

  else if (tipo == ERROR){
    perror("Erro na alocação. Tente novamente e cheque a entrada.");
    exit(1);
  }

  arquivoBin1 = fopen(filename1, "rb");
  assert(arquivoBin1);
  arquivoBin2 = fopen(filename2, "rb");
  assert(arquivoBin2);
  fill_matrix(&matrix1, arquivoBin1);
  fill_matrix(&matrix2, arquivoBin2);
  int matrixSize = matrix3.height * matrix3.width;
  
  for (int c = 0; c < matrixSize; c += 1) {
    matrix3.h_rows[c] = 0;
  }
  print_matrix(matrix1, 'A');
  print_matrix(matrix2, 'B');
  print_matrix(matrix3, 'C');
  // Mark overall start time
  puts("Executing scalar_matrix_mult...");
  
  gettimeofday(&start, NULL);
  scalar_matrix_mult(scalar, &matrix1);
  gettimeofday(&stop, NULL);
  printf("Scalar Matrix Mult's time: %f ms\n",
         timedifference_msec(start, stop));
  print_matrix(matrix1, 'A');
  // Mark matrix matrix start time
  puts("Executing matrix_matrix_mult...");
  gettimeofday(&start, NULL);
  matrix_matrix_mult(&matrix1, &matrix2, &matrix3);
  gettimeofday(&stop, NULL);
  printf("Matrix Matrix Mult's time: %f ms\n",
         timedifference_msec(start, stop));
  FILE *arquivoBin3 = fopen(filename3, "wb");
  FILE *arquivoBin4 = fopen(filename4, "wb");
  fill_binary(&matrix1, arquivoBin3);
  fill_binary(&matrix3, arquivoBin4);
  fclose(arquivoBin1);
  fclose(arquivoBin2);
  fclose(arquivoBin3);
  fclose(arquivoBin4);
  print_matrix(matrix3, 'C');
  int maxError = 0;
  int diffError = 0;
  for (int a = 0; a < matrix3.width * matrix3.height; a++)
    maxError =
        (maxError > (diffError = fabs((double)(matrix3.h_rows[a] - (20480)))))
            ? maxError
            : diffError;
  printf("erros com thread = %d\n", maxError);
  free(matrix1.h_rows);
  free(matrix2.h_rows);
  free(matrix3.h_rows);
  hipFree(&matrix1.d_rows);
  hipFree(&matrix2.d_rows);
  hipFree(&matrix3.d_rows);
  // Mark overall stop time
  gettimeofday(&overall_t2, NULL);
  // Show elapsed overall time
  diferencaTotal = timedifference_msec(overall_t1, overall_t2);
  printf("Overall time: %.2f ms\n", diferencaTotal);

  return 0;
}

void fill_matrix(struct matrix *matrixX, FILE *arquivoBinX) {
  float *matrixRows;
  float valorLido;
  int aux;
  for (int i = 0; i < (matrixX->width * matrixX->height); i++) {
    matrixRows = matrixX->h_rows + i;
    aux = fread(&valorLido, sizeof(float), 1, arquivoBinX);
    if (aux == 0) {
      fprintf(stderr, "error reading file\n");
      exit(1);
    }
    *matrixRows = valorLido;
  }
}

void fill_binary(struct matrix *matrixX, FILE *arquivoBinX) {
  float *matrixRows;
  int aux;
  for (int i = 0; i < (matrixX->width * matrixX->height); i++) {
    matrixRows = matrixX->h_rows + i;
    // printf("%f\n", *matrixRows);
    aux = fwrite(matrixRows, sizeof(float), 1, arquivoBinX);
    if (aux == 0) {
      fprintf(stderr, "error reading file\n");
      exit(1);
    }
  }
}

void print_matrix(struct matrix matrix, char matrixChar) {
  printf("------------- Matrix %c -------------", matrixChar);
  for (int c = 0; c < matrix.height * matrix.width; c++) {
    if (c % 16 == 0) {
      putchar('\n');
    }
    if (c == 256) {
      puts("Ooops...256 printing limit found...skipping printing...");
      break;
    }
    printf("%.2f ", matrix.h_rows[c]);
  }
  putchar('\n');
  return;
}
